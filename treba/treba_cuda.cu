#include "hip/hip_runtime.h"
/**************************************************************************/
/*   treba - probabilistic FSM and HMM training and decoding              */
/*   Copyright © 2013 Mans Hulden                                         */

/*   This file is part of treba.                                          */

/*   Treba is free software: you can redistribute it and/or modify        */
/*   it under the terms of the GNU General Public License version 2 as    */
/*   published by the Free Software Foundation.                           */

/*   Treba is distributed in the hope that it will be useful,             */
/*   but WITHOUT ANY WARRANTY; without even the implied warranty of       */
/*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the        */
/*   GNU General Public License for more details.                         */

/*   You should have received a copy of the GNU General Public License    */
/*   along with treba.  If not, see <http://www.gnu.org/licenses/>.       */
/**************************************************************************/

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

#define UDIV_UP(a, b) (((a) + (b) - 1) / (b))
#define ALIGN_UP(a, b) (UDIV_UP(a, b) * (b))

#define CudaCheckError() __cudaCheckError(__FILE__,__LINE__)
inline void __cudaCheckError(const char *file, const int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(-1);
    }
}

struct gibbs_state_chain {
    int state;
    int sym;
};

/* Macro for accessing counts on host */
/* FSM */
#define Ccurr(SOURCE_STATE, SYMBOL, TARGET_STATE) (*((gibbs_counts) + (num_states * alphabet_size * (SOURCE_STATE) + (SYMBOL) * num_states + (TARGET_STATE))))

/* HMM */
#define CcurrHMMemit(STATE, SYMBOL) (*((gibbs_counts_emit) + (alphabet_size * (STATE) + (SYMBOL))))
#define CcurrHMMtrans(SOURCE_STATE, TARGET_STATE) (*((gibbs_counts_trans) + (num_states * (SOURCE_STATE) + (TARGET_STATE))))

/* Macro for accessing current counts on CUDA device */
/* FSM */
#define Ccurrdevice(SOURCE_STATE, SYMBOL, TARGET_STATE) (((d_gibbs_counts) + (dc_num_states * dc_alphabet_size * (SOURCE_STATE) + (SYMBOL) * dc_num_states + (TARGET_STATE))))

/* HMM */
#define CcurrdeviceEMIT(STATE, SYMBOL) ((d_gibbs_counts_emit + (dc_alphabet_size * (STATE) + (SYMBOL))))
#define CcurrdeviceTRANS(SOURCE, TARGET) ((d_gibbs_counts_trans + (dc_num_states * (SOURCE) + (TARGET))))

__global__ void init_random_generator(hiprandState *state, unsigned long seed) {
    int id;
    id = blockIdx.x * blockDim.x + threadIdx.x;

    /* From the hiprand manual 4.2, p. 20: */
    /* State setup can be an expensive operation. One way to speed up the setup is to use different seeds */
    /* for each thread and a constant sequence number of 0. This can be especially helpful if many        */ 
    /* generators need to be created. While faster to set up, this method provides less guarantees about  */
    /* the mathematical properties of the generated sequences. If there happens to be a bad interaction   */
    /* between the hash function that initializes the generator state from the seed and the periodicity   */ 
    /* of the generators, there might be threads with highly correlated outputs for some seed values.     */
    /* We do not know of any problem values; if they do exist they are likely to be rare.                 */

    /* In other words, the slow method is to use the same seed but different sequence numbers (id):       */
    /* hiprand_init(seed, id, 0, &state[id]);                                                              */
    /* Here, we trust their word that problem values are rare and use the (much) faster setup:            */
    hiprand_init(seed+id, 0, 0, &state[id]);
}

/* Update sampled (global) counts, i.e. add current counts to sampled counts */
__global__ void gibbs_sampler_update_kernel_fsm(int numthreads, unsigned int *d_gibbs_counts, unsigned int *d_gibbs_sampled_counts) {
    int id;
    id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numthreads) { return; }
    d_gibbs_sampled_counts[id] += d_gibbs_counts[id];
}

__global__ void gibbs_sampler_update_kernel_hmm(int numthreads, unsigned int *d_gibbs_counts_trans, unsigned int *d_gibbs_counts_emit, unsigned int *d_gibbs_sampled_counts_trans, unsigned int *d_gibbs_sampled_counts_emit, int transtablesize, int emittablesize) {
    int id;
    id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numthreads) { return; }
    if (id < transtablesize)
	d_gibbs_sampled_counts_trans[id] += d_gibbs_counts_trans[id];
    if (id < emittablesize)
	d_gibbs_sampled_counts_emit[id] += d_gibbs_counts_emit[id];
}

/* Constants (to avoid passing these as variables to the kernels) */
__constant__ int dc_num_states, dc_alphabet_size;
__constant__ unsigned int *dc_resamplable;
__constant__ float dc_beta, dc_beta_t, dc_beta_e, dc_ANbeta, *dc_weight_list;

/* Main Gibbs sampler kernels. Every thread chooses one new state k in the chain of */
/* observations proportional to the probability of the new chain that uses k.       */
/* We launch two kernels in series: one for choosing odd-numbered states, and       */
/* another for choosing even-numbered ones (since adjacent states depend on each    */
/* other, we can't sample _all_ states in parallel, only non-dependent ones.)       */

/* The process is the same for FSMs and HMMs:                                         */
/* (1) Calculate probabilities for all changes z_i -> k (for all possible k) at o_i   */
/* (2) Do a weighted selection based on the probabilities and change the state z_i to */
/*     the selected new state.                                                        */ 
/* (3) Change the transition (and emission for HMM) counts to reflect the new choice. */

__global__ void gibbs_sampler_kernel_fsm(int numthreads, struct gibbs_state_chain *mychain, unsigned int *d_gibbs_counts_states, unsigned int *d_gibbs_counts, hiprandState *globalrandstate, short int evenodd) {
    unsigned int id, a, aprev, z, zprev, znext, k, low, high, mid, idorig;
    float g_sum, g_k, ind1, ind2, ind0, cointoss;
    float *weight_ptr;
    id = idorig = blockIdx.x * blockDim.x + threadIdx.x;
    weight_ptr = dc_weight_list + id;

    if (id >= numthreads) { return; }
    id = id * 2 + evenodd;
    mychain = mychain + dc_resamplable[id];

    aprev = (mychain-1)->sym;    /* Previous symbol */
    zprev = (mychain-1)->state;  /* Previous state  */
    a = mychain->sym;            /* Current symbol  */
    z = mychain->state;          /* Current state   */
    znext = (mychain+1)->state;  /* Next state      */
    
    /* Find probabilities of changing current state z to some state k for all k */
    for (k = 0, g_sum = 0; k < dc_num_states; k++) {
	ind0 = ((zprev == k && aprev == a && znext == k) ? 1.0 : 0.0);
	ind1 = (z == k ? 1.0 : 0.0);
	ind2 = ((zprev == k && aprev == a && z == znext) ? 1.0 : 0.0);

	g_k = (((float)*Ccurrdevice(k,a,znext)) - ind1 -ind2 + dc_beta) * (((float)*Ccurrdevice(zprev,aprev,k)) - ind1 - ind2 + dc_beta + ind0) / ((float)d_gibbs_counts_states[k] - ind1 + dc_ANbeta);
	g_sum += g_k;
        weight_ptr[numthreads * k] = g_sum;
    }

    /* Do a binary search for the first element in weight_list    */
    /* larger than cointoss. This diverges, but is still slightly */
    /* faster than a linear (more coalesced) search.              */

    cointoss = hiprand_uniform(&globalrandstate[idorig]) * g_sum;

    for (low = 0, high = dc_num_states - 1; low != high; ) {
    	mid = (low + high) / 2;
    	if (weight_ptr[numthreads * mid] <= cointoss) {
    	    low = mid + 1;
    	} else {
    	    high = mid;
    	}
    }
    k = high;

    /* Update counts */
    if (k != z) {
	atomicAdd(Ccurrdevice(zprev,aprev,k),1);
	atomicSub(Ccurrdevice(zprev,aprev,z),1);
	atomicAdd(Ccurrdevice(k,a,znext),1);
	atomicSub(Ccurrdevice(z,a,znext),1);
	atomicAdd(&d_gibbs_counts_states[k],1);
	atomicSub(&d_gibbs_counts_states[z],1);
	mychain->state = k;	
    }
}

__global__ void gibbs_sampler_kernel_hmm(int numthreads, struct gibbs_state_chain *mychain, unsigned int *d_gibbs_counts_states, unsigned int *d_gibbs_counts_trans, unsigned int *d_gibbs_counts_emit, hiprandState *globalrandstate, short int evenodd) {
    unsigned int id, a, z, zprev, znext, k, low, high, mid, idorig;
    float g_sum, g_k, ind0, ind1, ind2, ind3, cointoss;
    float *weight_ptr;
    id = idorig = blockIdx.x * blockDim.x + threadIdx.x;
    weight_ptr = dc_weight_list + id;

    if (id >= numthreads) { return; }
    id = id * 2 + evenodd;
    mychain = mychain + dc_resamplable[id];

    zprev = (mychain-1)->state;  /* Previous state  */
    a = mychain->sym;            /* Current symbol  */
    z = mychain->state;          /* Current state   */
    znext = (mychain+1)->state;  /* Next state      */
    
    /* Find probabilities of changing current state z_i to some state k for all k */
    /* (except state 0 (INIT) and state n - 1 (END)                               */
    for (k = 1, g_sum = 0; k < dc_num_states - 1; k++) {

	ind0 = z == k ? 1.0 : 0.0;
	ind1 = (z == zprev && znext == k) ? 1.0 : 0.0;
	ind2 = (zprev == k && z == znext) ? 1.0 : 0.0;
	ind3 = (zprev == k && k == znext) ? 1.0 : 0.0;

        g_k = ((((float)*CcurrdeviceEMIT(k,a)) + dc_beta_e - ind0) / (((float)d_gibbs_counts_states[k]) - ind0 + dc_alphabet_size * dc_beta_e)) *
	      (((((float)*CcurrdeviceTRANS(zprev,k)) + dc_beta_t - ind0 - ind1) * (((float)*CcurrdeviceTRANS(k,znext)) - ind0 - ind2 + ind3 + dc_beta_t)) /
              (((float)d_gibbs_counts_states[k]) - ind0 + dc_num_states * dc_beta_t));

	g_sum += g_k;
        weight_ptr[numthreads * k] = g_sum;
    }

    /* Do a binary search for the first element in weight_list    */
    /* larger than cointoss. This diverges, but is still slightly */
    /* faster than a linear (more coalesced) search.              */

    cointoss = hiprand_uniform(&globalrandstate[idorig]) * g_sum;

    for (low = 1, high = dc_num_states - 2; low != high; ) {
    	mid = (low + high) / 2;
    	if (weight_ptr[numthreads * mid] <= cointoss) {
    	    low = mid + 1;
    	} else {
    	    high = mid;
    	}
    }
    k = high;

    /* Update current counts */

    if (k != z) {
	atomicAdd(CcurrdeviceTRANS(zprev,k),1);
	atomicSub(CcurrdeviceTRANS(zprev,z),1);

	atomicAdd(CcurrdeviceTRANS(k,znext),1);
	atomicSub(CcurrdeviceTRANS(z,znext),1);

	atomicAdd(CcurrdeviceEMIT(k,a),1);
	atomicSub(CcurrdeviceEMIT(z,a),1);

	atomicAdd(&d_gibbs_counts_states[k],1);
	atomicSub(&d_gibbs_counts_states[z],1);

	mychain->state = k;
    }
}

#define PROB double

struct wfsa {
    int num_states;
    int alphabet_size;
    PROB *state_table;
    PROB *final_table;
};

struct hmm {
    int num_states;
    int alphabet_size;
    PROB *transition_table;
    PROB *emission_table;
};

struct observations {
    int size;
    int *data;
    int occurrences;
    struct observations *next;
};

/* nvcc really is a C++ compiler (i.e. g++), so all interaction with C code needs to be declared extern "C" */

extern "C" {
    extern int g_alphabet_size, g_gibbs_burnin, g_maxiterations;
    extern struct gibbs_state_chain *gibbs_init_fsm(struct observations *o, int num_states, int alphabet_size, int *obslen);
    extern struct gibbs_state_chain *gibbs_init_hmm(struct observations *o, int num_states, int alphabet_size, int *obslen);
    double gibbs_sampler_cuda_fsm(struct wfsa *fsm, struct observations *o, double beta, int num_states, int maxiter, int burnin, int lag);
    double gibbs_sampler_cuda_hmm(struct hmm *hmm, struct observations *o, double beta_t, double beta_e, int num_states, int maxiter, int burnin, int lag);
    extern struct wfsa *gibbs_counts_to_wfsa(struct wfsa *fsm, unsigned int *gibbs_sampled_counts, unsigned int *gibbs_counts_sampled_states, int alphabet_size, int num_states, double beta, double ANbeta);
    extern struct hmm *gibbs_counts_to_hmm(struct hmm *hmm, unsigned int *gibbs_sampled_counts_trans, unsigned int *gibbs_sampled_counts_emit, unsigned int *gibbs_counts_sampled_states, int alphabet_size, int num_states, double beta_t, double beta_e);
    extern PROB loglikelihood_all_observations_fsm(struct wfsa *fsm, struct observations *o);
    extern PROB loglikelihood_all_observations_hmm(struct hmm *hmm, struct observations *o);
}

double gibbs_sampler_cuda_fsm(struct wfsa *fsm, struct observations *o, double beta, int num_states, int maxiter, int burnin, int lag) {
    int alphabet_size, obslen, i,j, bdimeven, tdimeven, bdimodd, tdimodd, budim, tudim, samplecount;
    struct gibbs_state_chain *chain;

    unsigned int *gibbs_counts, *gibbs_sampled_counts, *gibbs_counts_sampled_states, *gibbs_counts_states, *resamplable, numthreadseven, numthreadsodd, updatethreads, chainlength;
    unsigned int *d_gibbs_counts, *d_gibbs_sampled_counts, *d_gibbs_counts_states, *d_resamplable;
    float *d_weight_list, fANbeta, fbeta;
    double ANbeta;
    struct gibbs_state_chain *d_chain;

    hiprandState *d_random_state;

    hipSetDevice(0);
    /* We don't use shared memory, might as well use cache */
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    alphabet_size = g_alphabet_size + 1; /* Use extra symbol for end-of-word (#) */

    /* Init chain and counts locally */
    chain = gibbs_init_fsm(o, num_states, g_alphabet_size, &obslen);
    gibbs_counts = (unsigned int *) calloc(num_states * num_states * alphabet_size, sizeof(unsigned int));
    gibbs_sampled_counts = (unsigned int *) calloc(num_states * num_states * alphabet_size, sizeof(unsigned int));
    gibbs_counts_states = (unsigned int *) calloc(num_states, sizeof(unsigned int));
    gibbs_counts_sampled_states = (unsigned int *) calloc(num_states, sizeof(unsigned int));

    for (i = 0; i < obslen-1; i++) {
	Ccurr( (chain+i)->state , (chain+i)->sym, (chain+i+1)->state )++;
	gibbs_counts_states[(chain+i)->state]++;
    }
    resamplable = (unsigned int *) malloc(obslen * sizeof(unsigned int));
    /* Create array that indexes only the resamplable states              */
    /* That is, initial states (0 and states with incoming # not included */
    for (i = 0, j = 0; j < obslen; j++) {
	if (j == 0 || (chain+j-1)->sym == g_alphabet_size) { /* Don't resample "initial" states, i.e. first */
	    continue;                                        /* state in chain, or states preceded by #     */
	} else {
	    resamplable[i] = j;
	    i++;
	}
    }
    chainlength = i;

    //    fprintf(stderr, "CUDA: planning to launch %i threads\n", chainlength);

    /* Init constants */
    fbeta = (float)beta;
    fANbeta = alphabet_size * num_states * beta;
    hipMemcpyToSymbol(HIP_SYMBOL(dc_alphabet_size), &alphabet_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_num_states), &num_states, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_ANbeta), &fANbeta, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_beta), &fbeta, sizeof(float));

    /* Move chain and counts to device */
    hipMalloc(&d_gibbs_counts, num_states * num_states * alphabet_size * sizeof(unsigned int));
    hipMalloc(&d_gibbs_sampled_counts, num_states * num_states * alphabet_size * sizeof(unsigned int));
    hipMalloc(&d_gibbs_counts_states, num_states * sizeof(unsigned int));
    hipMalloc(&d_chain, obslen * sizeof(struct gibbs_state_chain));
    hipMalloc(&d_resamplable, chainlength * sizeof(unsigned int));
    hipMalloc(&d_weight_list, num_states * (chainlength / 2 + (chainlength % 2)) * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(dc_weight_list), &d_weight_list, sizeof(float *));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_resamplable), &d_resamplable, sizeof(unsigned int *));

    hipMemcpy(d_gibbs_counts, gibbs_counts, num_states * num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_sampled_counts, gibbs_sampled_counts, num_states * num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_counts_states, gibbs_counts_states, num_states * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_chain, chain, obslen * sizeof(struct gibbs_state_chain), hipMemcpyHostToDevice);
    hipMemcpy(d_resamplable, resamplable, chainlength * sizeof(unsigned int), hipMemcpyHostToDevice);

    /* We need to sample odd and even states in the chain separately to avoid */
    /* messing up the counts. So for each iteration we launch two consecutive */
    /* kernels, one to resample even-numbered states, and one to resample     */
    /* odd-numbered ones.                                                     */

    /* Even thread number = |chain|/2 + (|chain| % 2)  */
    /* Odd thread number = |chain|/2                   */
    /* Even threads access chain location: id*2        */
    /* Odd threads access chain location: id*2+1       */

    tdimeven = THREADS_PER_BLOCK;
    tdimodd = THREADS_PER_BLOCK;
    numthreadseven = chainlength/2 + (chainlength % 2);
    numthreadsodd =  chainlength/2 ;
    bdimeven = (int) ceil((double) numthreadseven / (double) tdimeven);
    bdimodd = (int) ceil((double) numthreadsodd / (double) tdimeven);
        
    if (bdimeven < 2) { tdimeven = ALIGN_UP(numthreadseven, 32); }
    if (bdimodd < 2)  { tdimodd = ALIGN_UP(numthreadsodd, 32);   }

    //fprintf(stderr,"Dimensions: EVEN: [%i x %i] ODD: [%i x %i]\n",bdimeven,tdimeven, bdimodd, tdimodd);

    /* Number of threads to do total count updates */
    updatethreads = num_states * num_states * alphabet_size;

    tudim = THREADS_PER_BLOCK;
    budim = (int) ceil((double) updatethreads / (double) tudim);
    if (budim < 2) { tudim = ALIGN_UP(updatethreads, 32); }
    /* fprintf(stderr,"Update dimensions: [%i x %i]\n",budim,tudim); */
    /* fprintf(stderr,"Update dimensions EVEN: [%i x %i]\n",bdimeven,tdimeven); */
    /* fprintf(stderr,"Update dimensions ODD: [%i x %i]\n",bdimodd,tdimodd); */
    /* fprintf(stderr,"EVEN: %i ODD: %i\n",numthreadseven, numthreadsodd); */

    /* fprintf(stderr,"Initing random: %i x %i\n", bdimeven*2, tdimeven); */
    /* fflush(stdout); */
    hipMalloc(&d_random_state, (bdimeven * 2 * tdimeven) * sizeof(hiprandState));
    CudaCheckError();

    init_random_generator<<<bdimeven*2,tdimeven>>>(d_random_state, time(NULL));
    CudaCheckError();
   
    for (i = 0, samplecount = 0; i < g_maxiterations; i++) {
	gibbs_sampler_kernel_fsm<<<bdimeven,tdimeven>>>(numthreadseven, d_chain, d_gibbs_counts_states, d_gibbs_counts, d_random_state, 0);
	CudaCheckError();
	hipDeviceSynchronize();
	gibbs_sampler_kernel_fsm<<<bdimodd,tdimodd>>>(numthreadsodd, d_chain, d_gibbs_counts_states, d_gibbs_counts, d_random_state, 1);
	CudaCheckError();
	hipDeviceSynchronize();
	if (i > burnin && (i - burnin) % lag == 0) {
	    gibbs_sampler_update_kernel_fsm<<<budim,tudim>>>(updatethreads, d_gibbs_counts, d_gibbs_sampled_counts);
	    CudaCheckError();
	    hipDeviceSynchronize();
	    samplecount++;
	}
	/* For parallel computing tests */
	if (i > 0 && (i == 10 || i == 100 || i % 1000 == 0)) {
	    if (i == 10 || i == 100)
		lag = 10;
	    burnin = i;

	    //fprintf(stderr, "Iteration: %i  Samples collected: %i\n", i, samplecount);
	    hipMemcpy(gibbs_sampled_counts, d_gibbs_sampled_counts, num_states * num_states * alphabet_size * sizeof(unsigned int), hipMemcpyDeviceToHost);	    
	    ANbeta = (double) (alphabet_size * num_states * beta);
	    fsm = gibbs_counts_to_wfsa(fsm, gibbs_sampled_counts, gibbs_counts_sampled_states, alphabet_size, num_states, beta, ANbeta);
	    fprintf(stderr, "%i\t%.17g\n", i, loglikelihood_all_observations_fsm(fsm, o));
	    for (j = 0; j <  num_states * num_states * alphabet_size; j++) {
		gibbs_sampled_counts[j] = 0;
	    }
	    hipMemcpy(d_gibbs_sampled_counts, gibbs_sampled_counts, num_states * num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
	}
    }

    /* Move collected counts back to host mem */
    hipMemcpy(gibbs_sampled_counts, d_gibbs_sampled_counts, num_states * num_states * alphabet_size * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Build WFSA from collected counts */
    ANbeta = (double) (alphabet_size * num_states * beta);
    fsm = gibbs_counts_to_wfsa(fsm, gibbs_sampled_counts, gibbs_counts_sampled_states, alphabet_size, num_states, beta, ANbeta);
    
    hipFree(d_gibbs_counts);
    hipFree(d_gibbs_counts_states);
    hipFree(d_chain);
    hipFree(d_resamplable);
    hipFree(d_weight_list);
    hipFree(d_random_state);
    hipDeviceReset();
    return(loglikelihood_all_observations_fsm(fsm, o));
}

double gibbs_sampler_cuda_hmm(struct hmm *hmm, struct observations *o, double beta_t, double beta_e, int num_states, int maxiter, int burnin, int lag) {
    int alphabet_size, obslen, i,j, bdimeven, tdimeven, bdimodd, tdimodd, budim, tudim, samplecount;
    struct gibbs_state_chain *chain;

    unsigned int *gibbs_counts_trans, *gibbs_counts_emit, *gibbs_sampled_counts_trans, *gibbs_sampled_counts_emit, *gibbs_counts_sampled_states, *gibbs_counts_states, *resamplable, numthreadseven, numthreadsodd, updatethreads, chainlength;
    unsigned int *d_gibbs_counts_trans, *d_gibbs_counts_emit, *d_gibbs_sampled_counts_trans, *d_gibbs_sampled_counts_emit, *d_gibbs_counts_states, *d_resamplable;
    float *d_weight_list, fbeta_t, fbeta_e;
    struct gibbs_state_chain *d_chain;

    hiprandState *d_random_state;

    alphabet_size = g_alphabet_size;
    hipSetDevice(0);
    /* We don't use shared memory, might as well use cache */
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    /* Init chain and counts locally */
    chain = gibbs_init_hmm(o, num_states, g_alphabet_size, &obslen);
    gibbs_counts_trans = (unsigned int *) calloc(num_states * num_states, sizeof(unsigned int));
    gibbs_counts_emit = (unsigned int *) calloc(num_states * alphabet_size, sizeof(unsigned int));
    gibbs_sampled_counts_trans = (unsigned int *) calloc(num_states * num_states, sizeof(unsigned int));
    gibbs_sampled_counts_emit = (unsigned int *) calloc(num_states * alphabet_size, sizeof(unsigned int));
    gibbs_counts_states = (unsigned int *) calloc(num_states, sizeof(unsigned int));
    gibbs_counts_sampled_states = (unsigned int *) calloc(num_states, sizeof(unsigned int));

    for (i = 0; i < obslen-1; i++) {
	CcurrHMMtrans( (chain+i)->state, (chain+i+1)->state )++;
	gibbs_counts_states[(chain+i)->state]++;
    }
    for (i = 0; i < obslen-1; i++) {
      if ((chain+i)->sym >= 0) {
	  CcurrHMMemit( (chain+i)->state, (chain+i)->sym)++;
      }
    }
    resamplable = (unsigned int *) malloc(obslen * sizeof(unsigned int));
    /* Create array that indexes only the resamplable states              */
    /* That is, initial states (0 and states with incoming # not included */
    for (i = 0, j = 0; j < obslen; j++) {
	if ((chain+j)->sym < 0)
	    continue;            /* Don't resample INIT or END states */
	resamplable[i] = j;
	i++;
    }
    chainlength = i;

    //fprintf(stderr, "CUDA: planning to launch %i threads\n", chainlength);

    /* Init constants */
    fbeta_t = (float)beta_t;
    fbeta_e = (float)beta_e;

    hipMemcpyToSymbol(HIP_SYMBOL(dc_alphabet_size), &alphabet_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_num_states), &num_states, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_beta_t), &fbeta_t, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_beta_e), &fbeta_e, sizeof(float));

    /* Move chain and counts to device */
    hipMalloc(&d_gibbs_counts_trans, num_states * num_states * sizeof(unsigned int));
    hipMalloc(&d_gibbs_counts_emit, num_states * alphabet_size * sizeof(unsigned int));
    hipMalloc(&d_gibbs_sampled_counts_trans, num_states * num_states * sizeof(unsigned int));
    hipMalloc(&d_gibbs_sampled_counts_emit, num_states * alphabet_size * sizeof(unsigned int));
    hipMalloc(&d_gibbs_counts_states, num_states * sizeof(unsigned int));
    hipMalloc(&d_chain, obslen * sizeof(struct gibbs_state_chain));
    hipMalloc(&d_resamplable, chainlength * sizeof(unsigned int));
    hipMalloc(&d_weight_list, num_states * (chainlength / 2 + (chainlength % 2)) * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(dc_weight_list), &d_weight_list, sizeof(float *));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_resamplable), &d_resamplable, sizeof(unsigned int *));

    hipMemcpy(d_gibbs_counts_trans, gibbs_counts_trans, num_states * num_states * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_counts_emit, gibbs_counts_emit, num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_sampled_counts_trans, gibbs_sampled_counts_trans, num_states * num_states * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_sampled_counts_emit, gibbs_sampled_counts_emit, num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_gibbs_counts_states, gibbs_counts_states, num_states * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_chain, chain, obslen * sizeof(struct gibbs_state_chain), hipMemcpyHostToDevice);
    hipMemcpy(d_resamplable, resamplable, chainlength * sizeof(unsigned int), hipMemcpyHostToDevice);

    /* We need to sample odd and even states in the chain separately to avoid */
    /* messing up the counts. So for each iteration we launch two consecutive */
    /* kernels, one to resample even-numbered states, and one to resample     */
    /* odd-numbered ones.                                                     */

    /* Even thread number = |chain|/2 + (|chain| % 2)  */
    /* Odd thread number = |chain|/2                   */
    /* Even threads access chain location: id*2        */
    /* Odd threads access chain location: id*2+1       */

    tdimeven = THREADS_PER_BLOCK;
    tdimodd = THREADS_PER_BLOCK;
    numthreadseven = chainlength/2 + (chainlength % 2);
    numthreadsodd =  chainlength/2 ;
    bdimeven = (int) ceil((double) numthreadseven / (double) tdimeven);
    bdimodd = (int) ceil((double) numthreadsodd / (double) tdimeven);
        
    if (bdimeven < 2) { tdimeven = ALIGN_UP(numthreadseven, 32); }
    if (bdimodd < 2)  { tdimodd = ALIGN_UP(numthreadsodd, 32);   }

    //fprintf(stderr,"Dimensions: EVEN: [%i x %i] ODD: [%i x %i]\n",bdimeven,tdimeven, bdimodd, tdimodd);

    /* Number of threads to do total count updates */
    updatethreads = num_states * (num_states > alphabet_size ? num_states : alphabet_size);

    tudim = THREADS_PER_BLOCK;
    budim = (int) ceil((double) updatethreads / (double) tudim);
    if (budim < 2) { tudim = ALIGN_UP(updatethreads, 32); }
    //fprintf(stderr,"Update dimensions: [%i x %i]\n",budim,tudim);
    hipMalloc(&d_random_state, (bdimeven * 2 * tdimeven) * sizeof(hiprandState));
    CudaCheckError();
    init_random_generator<<<bdimeven*2,tdimeven>>>(d_random_state, time(NULL));
   
    for (i = 0, samplecount = 0; i < g_maxiterations; i++) {
	gibbs_sampler_kernel_hmm<<<bdimeven,tdimeven>>>(numthreadseven, d_chain, d_gibbs_counts_states, d_gibbs_counts_trans, d_gibbs_counts_emit, d_random_state, 0);
	hipDeviceSynchronize();
	gibbs_sampler_kernel_hmm<<<bdimodd,tdimodd>>>(numthreadsodd, d_chain, d_gibbs_counts_states, d_gibbs_counts_trans, d_gibbs_counts_emit, d_random_state, 1);
	hipDeviceSynchronize();
	if (i > burnin && (i - burnin) % lag == 0) {
	    gibbs_sampler_update_kernel_hmm<<<budim,tudim>>>(updatethreads, d_gibbs_counts_trans, d_gibbs_counts_emit, d_gibbs_sampled_counts_trans, d_gibbs_sampled_counts_emit, num_states * num_states, num_states * alphabet_size);
	    hipDeviceSynchronize();
	    samplecount++;
	}
	if (i > 0 && i % 100 == 0) {
	    //fprintf(stderr, "Iteration: %i  Samples collected: %i\n", i, samplecount);
	}
	/* For parallel computing tests */
	if (i > 0 && (i == 10 || i == 100 || i % 1000 == 0)) {
	    if (i == 10 || i == 100)
		lag = 10;
	    burnin = i;

	    //fprintf(stderr, "Iteration: %i  Samples collected: %i\n", i, samplecount);
	    hipMemcpy(gibbs_sampled_counts_trans, d_gibbs_sampled_counts_trans, num_states * num_states * sizeof(unsigned int), hipMemcpyDeviceToHost);
	    hipMemcpy(gibbs_sampled_counts_emit, d_gibbs_sampled_counts_emit, num_states * alphabet_size * sizeof(unsigned int), hipMemcpyDeviceToHost);

	    hmm = gibbs_counts_to_hmm(hmm, gibbs_sampled_counts_trans, gibbs_sampled_counts_emit, gibbs_counts_sampled_states, alphabet_size, num_states, beta_t, beta_e);
	    fprintf(stderr, "%i\t%.17g\n", i, loglikelihood_all_observations_hmm(hmm, o));
	    for (j = 0; j <  num_states * num_states; j++) {
		gibbs_sampled_counts_trans[j] = 0;
	    }
	    for (j = 0; j <  num_states * alphabet_size; j++) {
		gibbs_sampled_counts_emit[j] = 0;
	    }
	    hipMemcpy(d_gibbs_sampled_counts_trans, gibbs_sampled_counts_trans, num_states * num_states * sizeof(unsigned int), hipMemcpyHostToDevice);
	    hipMemcpy(d_gibbs_sampled_counts_emit, gibbs_sampled_counts_emit, num_states * alphabet_size * sizeof(unsigned int), hipMemcpyHostToDevice);
	}
    }

    /* Move collected counts back to host mem */
    hipMemcpy(gibbs_sampled_counts_trans, d_gibbs_sampled_counts_trans, num_states * num_states * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(gibbs_sampled_counts_emit, d_gibbs_sampled_counts_emit, num_states * alphabet_size * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Build HMM from collected counts */
    hmm = gibbs_counts_to_hmm(hmm, gibbs_sampled_counts_trans, gibbs_sampled_counts_emit, gibbs_counts_sampled_states, alphabet_size, num_states, beta_t, beta_e);
    
    hipFree(d_gibbs_counts_trans);
    hipFree(d_gibbs_counts_emit);
    hipFree(d_gibbs_counts_states);
    hipFree(d_chain);
    hipFree(d_resamplable);
    hipFree(d_weight_list);
    hipFree(d_random_state);
    hipDeviceReset();
    return(loglikelihood_all_observations_hmm(hmm, o));
}
